#include "hip/hip_runtime.h"
/*****************************

WRITTEN BY ISTVÁN MÁRTON

*****************************/

#include<stdio.h>
#include<math.h>
#include<stdlib.h>
#include "functions.h"

__device__ void calc_jmin_jmax(int* index, unsigned long long int* jMin, unsigned long long int* jMax, unsigned long long int* steps, unsigned long long int* steps_remainder){
	*jMax = (*index + 1) * *steps - 1;
	*jMin = *index * *steps;
	if(*index < *steps_remainder) *jMax += *index + 1;
	else *jMax += *steps_remainder;
	if(*index <= *steps_remainder) *jMin += *index;
	else *jMin += *steps_remainder;
}

__global__ void L1(int_type* d_mtx_as_vec, unsigned long long int steps, unsigned long long int steps_remainder, int_type *d_L1_vector, int *d_L1_strategy, int iShorter, int iLonger){ // This function calculates the L1 norm.
	int i, l, index, vect[NUM_OF_BITS - 1];
	int_type temp[length], product, L1;
	unsigned long long int number, jMax, jMin, iNumofZeros, aux;

	index = blockIdx.x * blockDim.x + threadIdx.x; // Index of threads.
	calc_jmin_jmax(&index, &jMin, &jMax, &steps, &steps_remainder); // This function calculates the minimal (jMin-th) and the maximal (jMax-th) word of the binary reflected Gray code for which the calculations must be performed by a given thread.

	number = jMin;
	for(l=0; l < iLonger; l++) {temp[l] = d_mtx_as_vec[(iShorter - 1) * iLonger + l];} // As the code can consider a row of the matrix with a fixed sign, it considers the last row of the matrix with +1.
	product = 0;
	for(i = 0 ; (iShorter - 1) > i; i++){
		iNumofZeros=(unsigned long long int) 1 << i;
		vect[i] = ((number+ iNumofZeros) >> (i+1)) & 1; // floor((j + 2^i)/2^(i+1)) Logical can be 0 and 1. logical is the number-th word and i-th digit of the BRGC.
		if(vect[i] == 1){for(l=0; l < iLonger; l++){temp[l] += d_mtx_as_vec[i * iLonger + l]; }} // The code determines the vector-matrix multiplication belonging to the number-th word of the BRGC.
		else {for(l=0; l < iLonger; l++){temp[l] -= d_mtx_as_vec[i * iLonger + l]; }}				
	}
	for(l= 0; l < iLonger; l++) {product += abs(temp[l]);} // The code calculates the L1 Bell value belonging to the number-th word of the BRGC.
	L1 = product; 
	for(l=0; l< (iShorter - 1); l++){d_L1_strategy[index * (iShorter - 1) + l] = vect[l];} // The program stores the strategy vector belonging to the number-th BRGC word in the d_L1_strategy vector.

	for(number=jMin + 1; number <= jMax; number++){ //The code determines the BRGC words until number variable reaches jMax.
		product = 0;
		aux = number;
		for(i = 0; (aux & 1) == 0; i++){
			aux = aux >> 1;
		}
		if(vect[i] == 0){vect[i] = 1; for(l=0; l < iLonger; l++){temp[l] += 2 * d_mtx_as_vec[i * iLonger + l]; product += abs(temp[l]);}} // When the i-th digit is changed, the code changes the result of the vector-matrix multiplication. It only needs to deal with the i-th row of the matrix.
		else {vect[i] = 0; for(l=0; l < iLonger; l++){temp[l] -= 2 * d_mtx_as_vec[i * iLonger + l]; product += abs(temp[l]);}}
		if(product > L1) {
			L1 = product; // If the current L1 sum, stored in product, is greater than the previous one, it modifies both the value 
			for(l=0; l<(iShorter - 1); l++){d_L1_strategy[index * (iShorter - 1) + l] = vect[l];} // and the corresponding strategy vector as well.
		}
	}
	d_L1_vector[index] = L1; // Every thread writes the biggest found L1 sum to d_L1_vector.
}

__global__ void L2(int_type* d_mtx_as_vec, unsigned long long int steps, unsigned long long int steps_remainder, int_type *d_L2_vector, int *d_L2_strategy, int iRows, int iCols){ // This function calculates the L2 norm.
	int i, l, index, vect[NUM_OF_BITS - 1];
	int_type temp_0[length], temp_1[length], product, L2;
	unsigned long long int number, jMax, jMin, iNumofZeros, aux;

	index = blockIdx.x * blockDim.x + threadIdx.x; // Index of threads.

	calc_jmin_jmax(&index, &jMin, &jMax, &steps, &steps_remainder); // This function calculates the minimal (jMin-th) and the maximal (jMax-th) word of the binary reflected Gray code for which the calculations must be performed by a given thread.
	
	number = jMin;
	for(l=0; l < iCols; l++) {temp_0[l] = d_mtx_as_vec[(iRows-1) * iCols + l]; temp_1[l] = 0; } // As the code can consider a row of the matrix with a fixed label, it considers the last row of the matrix with 0.
	product = 0;
	for(i = 0 ; (iRows-1) > i; i++){
		iNumofZeros=(unsigned long long int) 1 << i;
		vect[i] = ((number+ iNumofZeros) >> (i+1)) & 1; // floor((j + 2^i)/2^(i+1)) Logical can be 0 and 1. logical is the number-th word and i-th digit of the BRGC.
			if(vect[i] == 1){for(l=0; l < iCols; l++){temp_1[l] += d_mtx_as_vec[i * iCols + l]; }}
			else {for(l=0; l < iCols; l++){temp_0[l] += d_mtx_as_vec[i * iCols + l]; }}				
	}

	for(l= 0; l < iCols; l++) {product += abs(temp_0[l]) + abs(temp_1[l]);}
	L2 = product; // The code calculates the first L2 sum associated with the thread.
	for(l=0; l<(iRows - 1); l++){d_L2_strategy[index * (iRows - 1) + l] = vect[l];} // The code writes the first possible strategy vector into the d_L2_strategy vector.

	for(number=jMin + 1; number <= jMax; number++){
		product = 0;
		aux = number;
		for(i = 0; (aux & 1) == 0; i++){
			aux = aux >> 1;
		}
		if(vect[i] == 0){vect[i]=1; for(l=0; l < iCols; l++){temp_1[l] += d_mtx_as_vec[i * iCols + l]; temp_0[l] -= d_mtx_as_vec[i * iCols + l]; product += abs(temp_0[l]) + abs(temp_1[l]);}} // When the i-th digit is changed, the code changes the result defined by the definition of the L2 norm. It only needs to deal with the i-th row of the matrix.
		else {vect[i]=0; for(l=0; l < iCols; l++){temp_1[l] -= d_mtx_as_vec[i * iCols + l]; temp_0[l] += d_mtx_as_vec[i * iCols + l]; product += abs(temp_0[l]) + abs(temp_1[l]);}}	
		if(product > L2) {
			L2 = product; // If the current L2 sum is greater than the previous one, it modifies both the value
			for(l=0; l<(iRows - 1); l++){d_L2_strategy[index * (iRows - 1) + l] = vect[l];} // and the corresponding strategy vector as well.
		}
    }
	d_L2_vector[index] = L2; // Every thread writes the biggest found L2 sum to the d_L2_vector.
}

__global__ void L3(int_type* d_mtx_as_vec, unsigned long long int steps, unsigned long long int steps_remainder, int_type *d_L3_vector, int *d_L3_strategy, int iRows, int iCols, unsigned long long int *d_iNumPower){ // This function calculates the L3 norm.
	int i, l, index, vect[NUM_OF_BITS - 1], temporary, d_iPattern[6] = {0, 1, 2, 2, 1, 0}; // d_iPattern describes the pattern of the ternary reflected Gray code (TRGC).

	int_type temp_0[length], temp_1[length], temp_2[length], product, L3;
	unsigned long long int number, jMax, jMin, divide, logical;

	index = blockIdx.x * blockDim.x + threadIdx.x; // Index of threads.

	calc_jmin_jmax(&index, &jMin, &jMax, &steps, &steps_remainder); // This function calculates the minimal (jMin-th) and the maximal (jMax-th) word of the binary reflected Gray code for which the calculations must be performed by a given thread.
	
	number = jMin;
	for(l=0; l < iCols; l++) {temp_0[l] = d_mtx_as_vec[(iRows-1) * iCols + l]; temp_1[l] = 0; temp_2[l] = 0;} // As the code can consider a row of the matrix with a fixed label, it considers the last row of the matrix with 0.
	product = 0;
	for(i = 0 ; (iRows - 1) > i; i++){
		logical = (number/d_iNumPower[i]) % 6; // Determines the ternary reflected Gray code (TRGC). d_iNumPower is a vector consisting of the powers of 3.
		vect[i] = d_iPattern[logical]; // vect is the possible strategy vector. Its elements consist of 0, +1 or +2.
		switch(vect[i]) {
			case 0:
				for(l=0; l < iCols; l++){temp_0[l] += d_mtx_as_vec[i * iCols + l]; }
				break;
			case 1:
				for(l=0; l < iCols; l++){temp_1[l] += d_mtx_as_vec[i * iCols + l]; }
				break;
			case 2:
				for(l=0; l < iCols; l++){temp_2[l] += d_mtx_as_vec[i * iCols + l]; }
				break; // Every time the code finds the change in the TRGC, it stops searching for further changes.
		}				
	}
	
	for(l= 0; l < iCols; l++) {product += abs(temp_0[l]) + abs(temp_1[l]) + abs(temp_2[l]);}
	L3 = product; // The code calculates the first L3 sum associated with the thread.
	for(l=0; l<(iRows - 1); l++){d_L3_strategy[index * (iRows - 1) + l] = vect[l];} // The code writes the first possible strategy vector into the d_L3_strategy vector.

	for(number=jMin + 1; number <= jMax; number++){
      	product = 0;
		for(i = 0 ; (iRows - 1) > i; i++){
			divide = number/d_iNumPower[i];
			logical = divide % 3; // The code determines if there is a change in the i-th digit in the TRGC.
			if(logical) {
				logical = divide % 6; // If there is a change, the code calculates the value of the TRGC at that position.
				temporary = d_iPattern[logical];
				if( (vect[i] == 0)  && (temporary == 1) ) {for(l=0; l < iCols; l++){temp_0[l] -= d_mtx_as_vec[i * iCols + l]; temp_1[l] += d_mtx_as_vec[i * iCols + l]; }}// When the i-th digit is changed, the code changes the result defined by the definition of the L3 norm. It only needs to deal with the i-th row of the matrix.
				else if((vect[i] == 1)  && (temporary == 2)) {for(l=0; l < iCols; l++){temp_1[l] -= d_mtx_as_vec[i * iCols + l]; temp_2[l] += d_mtx_as_vec[i * iCols + l]; }}
				else if((vect[i] == 2)  && (temporary == 1)) {for(l=0; l < iCols; l++){temp_1[l] += d_mtx_as_vec[i * iCols + l]; temp_2[l] -= d_mtx_as_vec[i * iCols + l]; }}
				else {for(l=0; l < iCols; l++){temp_0[l] += d_mtx_as_vec[i * iCols + l]; temp_1[l] -= d_mtx_as_vec[i * iCols + l]; }}
				vect[i] = temporary;
				break; // Every time the code finds the change in the d-ary Gray code, it stops searching for further changes.
			}				
		}
		
		for(l= 0; l < iCols; l++) {product += abs(temp_0[l]) + abs(temp_1[l]) + abs(temp_2[l]);} // The code calculates the next L3 sum.
		if(product > L3) {
			L3 = product; // If the current L3 sum is greater than the previous one, it modifies both the value
			for(l=0; l<(iRows - 1); l++){d_L3_strategy[index * (iRows - 1) + l] = vect[l];} // and the corresponding strategy vector as well.
		}
	}
	d_L3_vector[index] = L3; // Every thread writes the biggest found L3 sum to the d_L3_vector.
}

__global__ void Ln(int_type* d_mtx_as_vec, int* d_iPattern, unsigned long long int steps, unsigned long long int steps_remainder, int_type *d_Ln_vector, int *d_Ln_strategy, int iRows, int iCols, int n, unsigned long long int *d_iNumPower){ // This function calculates the Ld norm.
	int i, l, index, vect[NUM_OF_BITS - 1], temporary;
	int_type temp[RANK_OF_NORM][length], product, Ln;
	unsigned long long int number, jMax, jMin, divide, logical;

	index = blockIdx.x * blockDim.x + threadIdx.x; // Index of threads.

	calc_jmin_jmax(&index, &jMin, &jMax, &steps, &steps_remainder); // This function calculates the minimal (jMin-th) and the maximal (jMax-th) word of the binary reflected Gray code for which the calculations must be performed by a given thread.
	
	number = jMin;
	for(l=0; l < iCols; l++) { // Initializes the temp variable.
		temp[0][l] = d_mtx_as_vec[(iRows-1) * iCols + l]; // As the code can consider a row of the matrix with a fixed label, it considers the last row of the matrix with 0.
		for(i=1; i<n; i++){
			temp[i][l] = 0;
		}
	}
	
	product = 0;
	for(i = 0 ; (iRows - 1) > i; i++){
		logical = (number/d_iNumPower[i]) % (2*n); 
		vect[i] = d_iPattern[logical]; //d_iPattern is a vector consisting of the powers of n. It helps determine the words of the n-ary Gray code.
		for(l=0; l < iCols; l++){temp[vect[i]][l] += d_mtx_as_vec[i * iCols + l]; }				
	}

	for(l= 0; l < iCols; l++) {
		for(i=0; i < n; i++){
			product += abs(temp[i][l]);
		}
	}
	Ln = product;  // The code calculates the first Ln sum associated with the thread.
	for(l=0; l<(iRows - 1); l++){d_Ln_strategy[index * (iRows - 1) + l] = vect[l];} // The code writes the first possible strategy vector into the d_Ln_strategy vector.

	for(number=jMin + 1; number <= jMax; number++){
		product = 0;
		for(i = 0 ; (iRows - 1) > i; i++){
			divide = number/d_iNumPower[i];
			logical = divide % n; // The code calculates if there is a change in the i-th digit in the n-ary Gray code.
			if(logical) {
				logical = divide % (2*n); // If there is a change, the code determines the value of the n-ary Gray code at that position.
				temporary = d_iPattern[logical];
				for(l=0; l < iCols; l++) {temp[vect[i]][l] -= d_mtx_as_vec[i * iCols + l]; temp[temporary][l] += d_mtx_as_vec[i * iCols + l]; } // When the i-th digit is changed, the code changes the result defined by the definition of the Ln norm. It only needs to deal with the i-th row of the matrix.
				vect[i] = temporary;
				break;
			}			
		}
		for(l= 0; l < iCols; l++) { // The code calculates the next Ln sum.
			for(i=0; i < n; i++){product += abs(temp[i][l]);}
		}
		if(product > Ln) {
			Ln = product; // If the current Ln sum is greater than the previous one, it modifies both the value
			for(l=0; l<(iRows - 1); l++){d_Ln_strategy[index * (iRows - 1) + l] = vect[l];} // and the corresponding strategy vector as well.
		}
	}
	d_Ln_vector[index] = Ln; // Every thread writes the biggest found Ln sum to the d_Ln_vector.
}

void calc_Parameters(item* first, item_calc* second, int* num_of_blocks, int* num_of_threads_per_block){

	if(first->n_original < second->iRows_reduced || first->n_original == 1) {second->n = first->n_original;}
	else{
		second->n = second->iRows_reduced > 1 ? second->iRows_reduced : 2;
		printf("The preprocessed matrix has a number of rows (%d) less than or equal to the order of the L norm (%d).\n",second->iRows_reduced ,first->n_original);
	}

	if(second->n == 1){
		first->total_num_to_calc = second->iRows_reduced == 0 ? 0 : (unsigned long long int) 1 << (second->iRows_reduced - 1); //The total number of L sums to be calculated is 2^(iRows_reduced-1)
		second->maxRows = NUM_OF_BITS - 1;
	}
	else if(second->n == 2){
		first->total_num_to_calc = second->iRows_reduced == 0 ? 0 : (unsigned long long int) 1 << (second->iRows_reduced - 1);
		second->maxRows = NUM_OF_BITS - 1;
	}
	else if(second->n == 3){
		first->total_num_to_calc = second->iRows_reduced == 0 ? 0 : pow(3, second->iRows_reduced - 1);
		second->maxRows = (int) (floor (NUM_OF_BITS / log2(second->n)) + 1);
	}
	else{
		first->total_num_to_calc = second->iRows_reduced == 0 ? 0 : pow(second->n, second->iRows_reduced - 1);
		second->maxRows = (int) (floor (NUM_OF_BITS / log2(second->n)) + 1);
	}

	*num_of_threads_per_block = *num_of_threads_per_block > first->total_num_to_calc ? first->total_num_to_calc : *num_of_threads_per_block;
	if(*num_of_threads_per_block == 0) {*num_of_blocks = 0;}
	else {*num_of_blocks = first->total_num_to_calc/ *num_of_threads_per_block > *num_of_blocks ? *num_of_blocks : first->total_num_to_calc/ *num_of_threads_per_block;}
	second->copyNum = *num_of_blocks * *num_of_threads_per_block;

	second->steps = second->copyNum == 0 ? 0 : first->total_num_to_calc/ second->copyNum;
	second->steps_remainder = second->copyNum == 0 ? 0 : first->total_num_to_calc % second->copyNum;

	printf("num_ofBlock: %d, num_ofThread in block: %d\n", *num_of_blocks, *num_of_threads_per_block);
	printf("maximum length of strategy vector: %d\n", second->maxRows);
	if(second->maxRows < second->iRows_reduced){printf("Matrix is too large. The matrix has %d rows after reduction. The maximum number of rows cannot be more than %d.\n", second->iRows_reduced, second->maxRows); exit(-1);}
	if(second->iCols_reduced > length) {printf("Matrix is too large. The length variable %d should be greater or equal than %d.\n", length, second->iCols_reduced); exit(-1);}
}

void calc_Lnorm(item_calc* second, int* num_of_blocks, int* num_of_threads_per_block){
	int i, iMax, *Ln_strategy, *d_Ln_strategy; //iMax: the index of the strategy vector found to be optimal; mtx_as_vec: the input matrix is converted to a vector in the host; d_mtx_as_vec: the converted matrix in the device; maxRows: them maximal number of rows (in the case of L1, the maximal number of rows or columns) of the matrix the program can deal with (this number is determined by the order of the L norm that should be calculated); Ln_vector and d_Ln_vector are the two vectors containing the possible L norms belonging to a given thread in the host and device, respectively; Ln_strategy and d_Ln_strategy are the two vector containing all of the possible strategy vectors belonging to different threads; num_ofBlock is the number of blocks the program uses; num_ofThread is the number of threads in a block
	int_type *d_mtx_as_vec, *Ln_vector, *d_Ln_vector;
	Ln_vector = (int_type*)calloc(second->copyNum, sizeof(int_type)); // The code allocates memory in the host for the possible L norms.
	Ln_strategy = (int*) malloc(second->copyNum * (second->iRows_reduced - 1) * sizeof(int)); // The code allocates memory for the strategies belonging to the possible L norms in the host.
	second->strategy = (int*) calloc(second->iRows_reduced - 1, sizeof(int));
	hipMalloc((void**)&d_mtx_as_vec, second->iRows_reduced * second->iCols_reduced * sizeof(int_type)); // Allocating memory for the matrix in the device.
	hipMalloc((void**)&d_Ln_vector, second->copyNum * sizeof(int_type)); // The code allocates memory in the device for the possible L norms.
	hipMalloc((void**)&d_Ln_strategy, second->copyNum * (second->iRows_reduced - 1) * sizeof(int)); // The code allocates memory for the strategies belonging to the possible L norms in the device.
	hipMemcpy(d_mtx_as_vec, second->mtx_as_vec, second->iRows_reduced * second->iCols_reduced * sizeof(int_type), hipMemcpyHostToDevice); // The matrix is copied from RAM to GPU memory.
	if(second->n == 1){ // If the order of the L norm is 1 then this part of the code will be executed.
		L1<<<*num_of_blocks, *num_of_threads_per_block>>>(d_mtx_as_vec, second->steps, second->steps_remainder, d_Ln_vector, d_Ln_strategy, second->iRows_reduced, second->iCols_reduced); // The calculation of the L1 norm with GPU.
	}
	else if(second->n == 2){ // If the order of the L norm is 2 then this part of the code will be executed.
		L2<<<*num_of_blocks, *num_of_threads_per_block>>>(d_mtx_as_vec, second->steps, second->steps_remainder, d_Ln_vector, d_Ln_strategy, second->iRows_reduced, second->iCols_reduced);
	}
	else if(second->n == 3){ // If the order of the L norm is 3 then this part of the code will be executed.
		unsigned long long int *iNumPower, *d_iNumPower;// iNumPower is copied to the device memory to speed up the calculation of the d-ary Gray code
		iNumPower = calc_iNumPower(second);
		hipMalloc((void**)&d_iNumPower, (second->maxRows-1) * sizeof(unsigned long long int));
		hipMemcpy(d_iNumPower, iNumPower, (second->maxRows-1) * sizeof(unsigned long long int), hipMemcpyHostToDevice);
		L3<<<*num_of_blocks, *num_of_threads_per_block>>>(d_mtx_as_vec, second->steps, second->steps_remainder, d_Ln_vector, d_Ln_strategy, second->iRows_reduced, second->iCols_reduced, d_iNumPower);
		free(iNumPower);
		hipFree(d_iNumPower);
	}
	else{ // If the order of the L norm is greater than 3, then this part of the code will be executed.
		unsigned long long int *iNumPower, *d_iNumPower;// iNumPower is copied to the device memory to speed up the calculation of the d-ary Gray code
		int *iPattern, *d_iPattern; // iPattern describes the d-ary Gray code.
		iPattern = calc_Pattern(&(second->n));
		iNumPower = calc_iNumPower(second);
		hipMalloc((void**)&d_iNumPower, (second->maxRows-1) * sizeof(unsigned long long int));
		hipMalloc((void**)&d_iPattern, 2 * second->n * sizeof(int));
		hipMemcpy(d_iNumPower, iNumPower, (second->maxRows-1) * sizeof(unsigned long long int), hipMemcpyHostToDevice);
		hipMemcpy(d_iPattern, iPattern, 2 * second->n * sizeof(int), hipMemcpyHostToDevice);
		Ln<<<*num_of_blocks, *num_of_threads_per_block>>>(d_mtx_as_vec, d_iPattern, second->steps, second->steps_remainder, d_Ln_vector, d_Ln_strategy, second->iRows_reduced, second->iCols_reduced, second->n, d_iNumPower);
		free(iNumPower);
		free(iPattern);
		hipFree(d_iNumPower);
		hipFree(d_iPattern);
	}
	hipMemcpy(Ln_vector, d_Ln_vector, second->copyNum * sizeof(int_type), hipMemcpyDeviceToHost); // Copy the possible L norm values from device to host.
	hipMemcpy(Ln_strategy, d_Ln_strategy, second->copyNum * (second->iRows_reduced - 1) * sizeof(int), hipMemcpyDeviceToHost); // Copy the possible strategies belonging to possible L norm values from device to host.
	second->Lnorm = Ln_vector[0];
	iMax = 0; // Determining the maximal element of Ln_vector, which is the L norm, and the index of the corresponding strategy vector as well.
	for(i = 1; i < second->copyNum; i++){ if(second->Lnorm < Ln_vector[i]) {second->Lnorm = Ln_vector[i]; iMax = i;}}
	for(i = 0; i < (second->iRows_reduced - 1); i++){second->strategy[i] = Ln_strategy[iMax * (second->iRows_reduced - 1) + i];}
	
	free(Ln_vector); // Deallocates the vectors in the host memory.
	free(Ln_strategy);
	
	hipFree(d_Ln_vector); // Deallocates the vectors in the device memory.
	hipFree(d_Ln_strategy);
	hipFree(d_mtx_as_vec);
}

void arguments_CUDA(item* first, int* num_of_blocks, int* num_of_threads_per_block, int* argc, char** argv, hipDeviceProp_t* devProp){
	FILE *fp;
	int sd, t;
	char msg[] = "Use the following command: ./L_CUDA number_of_blocks number_of_threads_per_block filename_of_matrix order_of_the_L_norm";
	if(*argc < 5){
		printf("Incorrect number of input arguments. %s\n", msg);
		exit(-1);
	}
	
	sd = sscanf(argv[1], "%d", &t);
	if((sd == 0) || (t < 1)){
		printf("Please make sure that the number of blocks is a positive integer. %s\n", msg);
		exit(-1);
	}
	*num_of_blocks = t;
	
	sd = sscanf(argv[2], "%d", &t);
	if((sd == 0) || (t < 1)){
		printf("Please make sure that the number of threads is a positive integer. %s\n", msg);
		exit(-1);
	}
	*num_of_threads_per_block = t;
	
	sprintf(first->fileName,"%s", argv[3]);
	fp = fopen(first->fileName, "r");
	if(fp == NULL){
		printf("Please make sure that the file containig the matrix exists within this directory. %s\n", msg);
	}
	fclose(fp);
	
	sd = sscanf(argv[4], "%d", &t);
	if((sd == 0) || (t < 1)){
		printf("Please make sure that the order of the L norm is a positive integer. %s\n", msg);
		exit(-1);
	}
	first->n_original = t;
	
	if(*argc < 6){first->stat = 'n';}
	else {first->stat = argv[5][0];}
	
	if(t > RANK_OF_NORM) {printf("The order of the L norm is too large. Please increase the RANK_OF_NORM variable in the code to %d and compile and run it again.\n", t); exit(-1);}
	if(*num_of_threads_per_block > devProp->maxThreadsPerBlock) {printf("The maximum number of threads per block cannot be more than %d.\n", devProp->maxThreadsPerBlock); exit(-1);}
}

void load_parameters(item* first, item_calc* second, int* argc, char** argv, hipDeviceProp_t* devProp, int* num_of_blocks, int* num_of_threads_per_block){
	arguments_CUDA(first, num_of_blocks, num_of_threads_per_block, argc, argv, devProp);
	matrix_read(first);
	eliminate_zero_rows_cols(first, second);
	if(first->stat == 'y' || first->stat == 'r' || first->stat == 'Y' || first->stat == 'R') {calc_reduce_matrix_rows(first, second); }
	if(first->stat == 'y' || first->stat == 'c' || first->stat == 'Y' || first->stat == 'C') {calc_reduce_matrix_cols(first, second); if(first->n_original > 1) {calc_reduce_matrix_cols_sign(first, second);} }
	if(first->stat == 'y' || first->stat == 'r' || first->stat == 'Y' || first->stat == 'R' || first->stat == 'c' || first->stat == 'C') { delete_rows(first, second); delete_cols(first, second);}
	if(first->n_original == 1) {convert_mtx_to_vec_Transpose(first, second); }
	else {convert_mtx_to_vec_noTranspose(first, second);}
	calc_Parameters(first, second, num_of_blocks, num_of_threads_per_block); // Calculates the necessary parameters for the calculation.
}

int main(int argc, char *argv[]){
	int num_of_threads_per_block, num_of_blocks;
	item first;
	item_calc second;
	hipDeviceProp_t devProp; // devProp contains the number of cores a warp contain.
	hipGetDeviceProperties(&devProp, 0);
	load_parameters(&first, &second, &argc, argv, &devProp, &num_of_blocks, &num_of_threads_per_block);
	if(second.iRows_reduced == 0) {printf("This is a zero matrix.\n"); second.Lnorm = 0;}
	else {calc_Lnorm(&second, &num_of_blocks, &num_of_threads_per_block);} // The function 'calc_Lnorm' calculates the L norm of order n of the input matrix.
	print_results(&first, &second);
	
	free_first(&first);
	if(second.iRows_reduced > 0) free_second(&second);
	return 0;
}
